#include "hip/hip_runtime.h"
/* kvec_cuda.cu: interface between C++ and CUDA for kvecs
 *
 * Written by Tom Grimes, 9-April 2010
 *
 * This call is blocking.
 */

// need k_struct
#include "ecpints.h"

#include "hip/hip_runtime.h"
#include "cutil_inline.h"

// kernel prototypes
__global__ void kvec_kernel
               (const double *CApows, const int dim_1, const int dim_2, const int dim_3,
                const k_struct *kvecs, const int nkvecs,
                const double *uniq_exp, 
                double *kvec_vals);
__global__ void kvp_kernel
               (const double *CApows, const int dim_1, const int dim_2, const int dim_3,
                const kvpair_struct *kvecs, const int nkvpairs,
                const double *uniq_exp,
                double *kvp_vals, double *screen);

// call this
void kvec_cuda(double *h_CApows, int nat, int maxdim,
               k_struct *h_kvecs, int nkvecs,
               kvpair_struct *h_kvpairs, int nkvpairs,
               double *h_exps, int nexp,
               double *kvec_vals,
               double *kvp_vals,
               double *scr_vals)
{
   // get device memory
   double *d_CApows, *d_exps, *d_kv_vals, *d_kvp_vals, *d_scr_vals;
   k_struct *d_kvecs;
   kvpair_struct *d_kvpairs;
   cutilSafeCall(hipMalloc((void**)&d_CApows, 3*nat*nat*maxdim*sizeof(double)));
   cutilSafeCall(hipMalloc((void**)&d_exps, nexp*sizeof(double)));
   cutilSafeCall(hipMalloc((void**)&d_kvecs, nkvecs*sizeof(k_struct)));
   cutilSafeCall(hipMalloc((void**)&d_kvpairs, nkvpairs*sizeof(kvpair_struct)));

   cutilSafeCall(hipMalloc((void**)&d_kv_vals, 4*nkvecs*sizeof(double)));
   cutilSafeCall(hipMalloc((void**)&d_kvp_vals, 4*nkvpairs*sizeof(double)));
   cutilSafeCall(hipMalloc((void**)&d_scr_vals, nkvpairs*sizeof(double)));

   // move input data to device
   cutilSafeCall(hipMemcpy(d_CApows, h_CApows, 3*nat*nat*maxdim*sizeof(double), hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(d_exps, h_exps, nexp*sizeof(double), hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(d_kvecs, h_kvecs, nkvecs*sizeof(k_struct), hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(d_kvpairs, h_kvpairs, nkvpairs*sizeof(kvpair_struct), hipMemcpyHostToDevice));

   // get block sizes
   const int nthread = 10;
   const int kv_blocks = nkvecs/nthread + (nkvecs%nthread==0 ? 0 : 1),
             kvp_blocks = nkvpairs/nthread + (nkvpairs%nthread==0 ? 0 : 1);

   // make the calls
   kvec_kernel<<<kv_blocks, nthread>>>(d_CApows, maxdim, 3*maxdim, nat*3*maxdim, d_kvecs, nkvecs, d_exps,
                                       d_kv_vals);
   cutilCheckMsg("kernel launch failure");

   kvp_kernel<<<kvp_blocks, nthread>>>(d_CApows, maxdim, 3*maxdim, nat*3*maxdim, d_kvpairs, nkvpairs, d_exps,
                                       d_kvp_vals, d_scr_vals);
   cutilCheckMsg("kernel launch failure");

   // copy information back to host
   cutilSafeCall(hipMemcpy(kvec_vals, d_kv_vals, 4*nkvecs*sizeof(double), hipMemcpyDeviceToHost));
   cutilSafeCall(hipMemcpy(kvp_vals, d_kvp_vals, 4*nkvpairs*sizeof(double), hipMemcpyDeviceToHost));
   cutilSafeCall(hipMemcpy(scr_vals, d_scr_vals, nkvpairs*sizeof(double), hipMemcpyDeviceToHost));

   // free device memory
   cutilSafeCall(hipFree(d_CApows));
   cutilSafeCall(hipFree(d_exps));
   cutilSafeCall(hipFree(d_kvecs));
   cutilSafeCall(hipFree(d_kvpairs));
   cutilSafeCall(hipFree(d_kv_vals));
   cutilSafeCall(hipFree(d_kvp_vals));
   cutilSafeCall(hipFree(d_scr_vals));

   return;
};

