#include "hip/hip_runtime.h"
/* rad_cuda.cu: interface between C++ and CUDA for radial integrals
 *
 * Written by Tom Grimes, 9-April 2010
 *
 * This call is blocking.
 */

#include "ecpints.h"

#include "hip/hip_runtime.h"
#include "cutil_inline.h"

// kernel prototypes
__global__ void local_radial_kernel
                             (const rad_int *rad_ints, const int *rad_list, 
                              const int lower, const int upper,
                              const double *kvp_vals, double *output);

__global__ void nonlocal_radial_kernel
                                (const rad_int *rad_ints, const int *rad_list,
                                 const int lower, const int upper,
                                 const double *kvec_vals, double *output);

// this function should be ELSEWHERE
void cuda_load_kvecs (double *h_kv_vals, int nkvecs, double *h_kvp_vals, int nkvpairs)
{
   cutilSafeCall(hipMemcpy(ecp_data.d_kvec_vals, h_kv_vals, 4*nkvecs*sizeof(double), hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(ecp_data.d_kvp_vals, h_kvp_vals, 4*nkvpairs*sizeof(double), hipMemcpyHostToDevice));
   return;
};

// call this
void radial_cuda(int lower, int upper, double *h_output)
{
   // get lower and upper limits for local and nonlocal
   int *local_ptr = &(ecp_data.c_local_rad.front()),
       *nonlocal_ptr = &(ecp_data.c_nonlocal_rad.front()),
       l_lower = lower_bound(local_ptr,local_ptr+ecp_data.c_local_rad.size(),lower)-local_ptr,
       l_upper = lower_bound(local_ptr,local_ptr+ecp_data.c_local_rad.size(),upper)-local_ptr,
       nl_lower = lower_bound(nonlocal_ptr,nonlocal_ptr+ecp_data.c_nonlocal_rad.size(),lower)-nonlocal_ptr,
       nl_upper = lower_bound(nonlocal_ptr,nonlocal_ptr+ecp_data.c_nonlocal_rad.size(),upper)-nonlocal_ptr;

   // get dimensions
   int nlocal = l_upper-l_lower,
       nnlocal = nl_upper-nl_lower,
       nlthreadx = 4,
       nlthready = 4,
       nlthread = nlthreadx*nlthready,
       nnlthreadx = 4,
       nnlthready = 4,
       nnlthread = nnlthreadx*nnlthready,
       nlblocks = nlocal/nlthread + (nlocal%nlthread==0? 0 : 1),
       nnlblocks = nnlocal/nnlthread + (nnlocal%nnlthread==0? 0 : 1),
       bdim1l, bdim2l, bdim1n, bdim2n;
   if (nlblocks>0)
   {
      bdim1l = (int) sqrt(nlblocks);
      bdim2l = nlblocks/bdim1l + (nlblocks%bdim1l==0? 0 : 1);
   };
   if (nnlblocks>0)
   {
      bdim1n = (int) sqrt(nnlblocks);
      bdim2n = nnlblocks/bdim1n + (nnlblocks%bdim1n==0? 0 : 1);
   };
   dim3 lblocks(bdim1l,bdim2l),
        nblocks(bdim1n,bdim2n),
        lthreads(nlthreadx,nlthready),
        nthreads(nnlthreadx,nnlthready);

   // make the calls
   if (l_upper>l_lower)
   {
      local_radial_kernel<<<lblocks,lthreads>>>(ecp_data.d_radints, ecp_data.d_local_rad,
                                               l_lower, l_upper,
                                               ecp_data.d_kvp_vals, ecp_data.d_radvals);
      cutilCheckMsg("kernel launch failure");
   };
   if (nl_upper>nl_lower)
   {
      nonlocal_radial_kernel<<<nblocks,nthreads>>>(ecp_data.d_radints, ecp_data.d_nonlocal_rad,
                                                  nl_lower, nl_upper,
                                                  ecp_data.d_kvec_vals, ecp_data.d_radvals);
      cutilCheckMsg("kernel launch failure");
   };

   // copy information back to host
   cutilSafeCall(hipMemcpy(h_output+lower, ecp_data.d_radvals+lower, (upper-lower)*sizeof(double),
                            hipMemcpyDeviceToHost));

   return;
};

