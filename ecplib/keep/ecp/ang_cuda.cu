#include "hip/hip_runtime.h"
/* ang_cuda.cu: interface between C++ and CUDA for angular integrals
 *
 * Written by Tom Grimes, 27-April 2010
 *
 * This call is blocking.
 */

#include "ecpints.h"

#include "hip/hip_runtime.h"
#include "cutil_inline.h"

// kernel prototypes
__global__ void local_angular_kernel
                             (const ang_struct *ang_ints, const int *ang_list,
                              const int lower, const int upper,
                              const double *kvp_vals,
                              const double *Ylm, const int max_l,
                              const double *local_tab, const int max_IJK,
                              double *output);

__global__ void nonlocal_angular_kernel
                             (const ang_struct *ang_ints, const int *ang_list,
                              const int lower, const int upper,
                              const double *kvec_vals,
                              const double *Ylm, const int max_l,
                              const double *nonlocal_tab, const int max_IJK,
                              double *output);

// call this
void angular_cuda(int lower, int upper, double *h_output)
{
   // get lower and upper limits for local and nonlocal angular ints
   int *local_ptr = &(ecp_data.c_local_ang.front()),
       *nonlocal_ptr = &(ecp_data.c_nonlocal_ang.front()),
       l_lower = lower_bound(local_ptr,local_ptr+ecp_data.c_local_ang.size(),lower)-local_ptr,
       l_upper = lower_bound(local_ptr,local_ptr+ecp_data.c_local_ang.size(),upper)-local_ptr,
       nl_lower = lower_bound(nonlocal_ptr,nonlocal_ptr+ecp_data.c_nonlocal_ang.size(),lower)-nonlocal_ptr,
       nl_upper = lower_bound(nonlocal_ptr,nonlocal_ptr+ecp_data.c_nonlocal_ang.size(),upper)-nonlocal_ptr;

   // get dimensions
   const int nlocal = l_upper-l_lower,
             nnonlocal = nl_upper-nl_lower,
             nlthreadx = 4,
             nlthready = 4,
             nlthreads = nlthreadx*nlthready,
             nnlthreadx = 16,
             nnlthready = 2,
             nlblocks = nlocal/nlthreads+(nlocal%nlthreads==0? 0 : 1),
             nnlblocks = nnonlocal/nnlthreadx+(nnonlocal%nnlthreadx==0? 0 : 1);
   int bdim1l = 0, bdim2l = 0, bdim1n = 0, bdim2n = 0;
   if (nlblocks>0)
   {
       bdim1l = (int) sqrt(nlblocks);
       bdim2l = nlblocks/bdim1l + (nlblocks%bdim1l==0? 0 : 1);
   };
   if (nnlblocks>0)
   {
       bdim1n = (int) sqrt(nnlblocks);
       bdim2n = nnlblocks/bdim1n + (nnlblocks%bdim1n==0? 0 : 1);
   };
   dim3 lblocks(bdim1l,bdim2l),
        nblocks(bdim1n,bdim2n),
        lthreads(nlthreadx,nlthready),
        nthreads(nnlthreadx,nnlthready);

   // make the calls
   if (nlocal>0)
   {
      local_angular_kernel<<<lblocks,lthreads>>>(ecp_data.d_angints, ecp_data.d_local_ang, l_lower, l_upper,
                                                 ecp_data.d_kvp_vals,
                                                 ecp_data.d_Ylm, ecp_data.Ylm_maxl,
                                                 ecp_data.d_l_ang, ecp_data.max_IJK,
                                                 ecp_data.d_angvals);
      cutilCheckMsg("kernel launch failure");
   };
   if (nnonlocal>0)
   {
      nonlocal_angular_kernel<<<nblocks,nthreads>>>(ecp_data.d_angints, ecp_data.d_nonlocal_ang, nl_lower, nl_upper,
                                                    ecp_data.d_kvec_vals,
                                                    ecp_data.d_Ylm, ecp_data.Ylm_maxl,
                                                    ecp_data.d_nl_ang, ecp_data.max_nl_IJK,
                                                    ecp_data.d_angvals);
      cutilCheckMsg("kernel launch failure");
   };

   // copy information back to host
   cutilSafeCall(hipMemcpy(h_output+lower, ecp_data.d_angvals+lower, (upper-lower)*sizeof(double),
                            hipMemcpyDeviceToHost));

   return;
};

