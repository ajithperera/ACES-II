/* ecp_init_cuda.cu: initialize storage on device
 *
 * Written by Tom Grimes, 13-April 2010
 *
 */

#include "ecpints.h"
#include "hip/hip_runtime.h"
#include "cutil_inline.h"

void ecp_init_cuda()
{
   // allocate device memory for radial ints
   int nradints=ecp_data.radints.size(),
       nkvecs=ecp_data.kvecs.size(),
       nkvp=ecp_data.kvpairs.size();
   cutilSafeCall(hipMalloc((void**)&ecp_data.d_radints, nradints*sizeof(rad_int)));
   cutilSafeCall(hipMalloc((void**)&ecp_data.d_kvec_vals, 4*nkvecs*sizeof(double)));
   cutilSafeCall(hipMalloc((void**)&ecp_data.d_kvp_vals, 4*nkvp*sizeof(double)));
   cutilSafeCall(hipMalloc((void**)&ecp_data.d_radvals, nradints*sizeof(double)));

   // transfer the radial int list
   cutilSafeCall(hipMemcpy(ecp_data.d_radints, &(ecp_data.radints.front()),
                            nradints*sizeof(rad_int), hipMemcpyHostToDevice));

   // get local/nonlocal ordering of radial ints
   ecp_data.c_local_rad.clear();
   ecp_data.c_nonlocal_rad.clear();
   for (int i=0; i<ecp_data.radints.size(); i++)
   {
      if (ecp_data.radints[i].lambdaP==-1)
      {
         ecp_data.c_local_rad.push_back(i);
      } else {
         ecp_data.c_nonlocal_rad.push_back(i);
      };
   };

   cutilSafeCall(hipMalloc((void**)&ecp_data.d_local_rad,ecp_data.c_local_rad.size()*sizeof(int)));
   cutilSafeCall(hipMalloc((void**)&ecp_data.d_nonlocal_rad,ecp_data.c_nonlocal_rad.size()*sizeof(int)));
   cutilSafeCall(hipMemcpy(ecp_data.d_local_rad, &(ecp_data.c_local_rad.front()),
                            ecp_data.c_local_rad.size()*sizeof(int), hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(ecp_data.d_nonlocal_rad, &(ecp_data.c_nonlocal_rad.front()),
                            ecp_data.c_nonlocal_rad.size()*sizeof(int), hipMemcpyHostToDevice));

   // transfer angular integral data
   int nangints = ecp_data.angints.size(),
       max_l = ecp_data.Ylm_maxl,
       nYlm = ((max_l+1)*(max_l+2)+1)*(max_l+1)*(max_l+1)*(max_l+1),
       max_IJK = ecp_data.max_IJK,
       nloctab = ((max_l+1)*(max_l+2)+1)*(max_IJK+1)*(max_IJK+1)*(max_IJK+1),
       max_nl_IJK = ecp_data.max_nl_IJK,
       nnltab = ((max_l+1)*(max_l+2)+1)*((max_l+1)*(max_l+2)+1)*(max_nl_IJK+1)*(max_nl_IJK+1)*(max_nl_IJK+1);
   cutilSafeCall(hipMalloc((void**)&ecp_data.d_angvals, nangints*sizeof(double)));
   cutilSafeCall(hipMalloc((void**)&ecp_data.d_angints, nangints*sizeof(ang_struct)));

   cutilSafeCall(hipMalloc((void**)&ecp_data.d_Ylm, nYlm*sizeof(double)));
   cutilSafeCall(hipMalloc((void**)&ecp_data.d_l_ang, nloctab*sizeof(double)));
   cutilSafeCall(hipMalloc((void**)&ecp_data.d_nl_ang, nnltab*sizeof(double)));
   cutilSafeCall(hipMemcpy(ecp_data.d_angints, &(ecp_data.angints.front()),
                 nangints*sizeof(ang_struct),hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(ecp_data.d_Ylm, ecp_data.Ylm,
                 nYlm*sizeof(double),hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(ecp_data.d_l_ang, ecp_data.local_ang,
                 nloctab*sizeof(double),hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(ecp_data.d_nl_ang, ecp_data.nonlocal_ang,
                 nnltab*sizeof(double),hipMemcpyHostToDevice));

   // separate local and nonlocal 
   ecp_data.c_local_ang.clear();
   ecp_data.c_nonlocal_ang.clear();
   for (int i=0; i<ecp_data.angints.size(); i++)
   {
      if (ecp_data.angints[i].ang2_lambda==-1)
      {
         ecp_data.c_local_ang.push_back(i);
      } else {
         ecp_data.c_nonlocal_ang.push_back(i);
      };
   };

   cutilSafeCall(hipMalloc((void**)&ecp_data.d_local_ang, ecp_data.c_local_ang.size()*sizeof(int)));
   cutilSafeCall(hipMalloc((void**)&ecp_data.d_nonlocal_ang, ecp_data.c_nonlocal_ang.size()*sizeof(int)));
   cutilSafeCall(hipMemcpy(ecp_data.d_local_ang, &(ecp_data.c_local_ang.front()),
                 ecp_data.c_local_ang.size()*sizeof(int),hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(ecp_data.d_nonlocal_ang, &(ecp_data.c_nonlocal_ang.front()),
                 ecp_data.c_nonlocal_ang.size()*sizeof(int),hipMemcpyHostToDevice));

   return;
};

